#include "hip/hip_runtime.h"
#include "Constants.h"

extern "C"{

    #define K1 607.760009765625f
    #define UR 36.459999084472656f
    #define K2 1260.56005859375f
    #define Kt 1.0f
    #define reflectanciaAtmosfera 0.029999999329447746f
    #define StefanBoltzman 5.669999936230852E-8f
    #define L 0.10000000149011612f
    #define julianDay 248.0f
    #define Uref 0.9207136034965515f
    #define Ta 32.7400016784668f
    #define P 99.30000305175781f
    #define Rg_24h 243.9499969482422f
    #define S 1367.0f
    #define latitude -16.559999465942383f
    #define Z 50.2400016784668f

    #define dr 0.985846566048888f
    #define cosZ 0.7687302359405433f
    #define declinacaoSolar 0.10110116629018565f
    #define anguloHorarioNascerSol 1.5406261742945568f
    #define rad_solar_toa 33.677489830352386f
    #define Rg_24h_mj 21.077279736328126f
    #define transmissividade24h 0.6258566134977164f
    #define ea 1.80713581697293f
    #define W 27.22280289964939f
    #define transmissividade 0.7299130836698269f
    #define emissivityAtm 0.7887452566722131f
    #define SWd 736.5564499520258f
    #define LWdAtm 391.54430119162527f

    __constant__ double parameterAlbedo[] = {0.2930000126361847f,0.27399998903274536f,0.2329999953508377f,0.15700000524520874f,0.032999999821186066f,0.0f,0.010999999940395355f};

    __constant__ double calibration1[] = {-1.5199999809265137,193.0,1957.0};
    __constant__ double calibration2[] = {-2.8399999141693115,365.0,1826.0};
    __constant__ double calibration3[] = {-1.1699999570846558,264.0,1554.0};
    __constant__ double calibration4[] = {-1.5099999904632568,221.0,1036.0};
    __constant__ double calibration5[] = {-0.3700000047683716,30.200000762939453,215.0};
    __constant__ double calibration6[] = {1.2378000020980835,15.303000450134277,1.0};
    __constant__ double calibration7[] = {-0.15000000596046448,16.5,80.66999816894531};

    __device__ void execute_sub(
        double pixel2,
        double pixel3,
        double pixel4,
        double pixel5,
        double pixel1,
        double pixel7,
        double pixel6,

        double * albedo,
        double * NDVI,
        double * SAVI,
        double * IAF,
        double * emissividadeNB,
        double * emissivity,
        double * Ts,
        double * LWd,
        double * Rn
    ){
        double sumBandas = 0;
        double banda1=calibration1[0]+((calibration1[1]-calibration1[0])/255.0f)*pixel1;
        double banda2=calibration2[0]+((calibration2[1]-calibration2[0])/255.0f)*pixel2;
        double banda3=calibration3[0]+((calibration3[1]-calibration3[0])/255.0f)*pixel3;
        double banda4=calibration4[0]+((calibration4[1]-calibration4[0])/255.0f)*pixel4;
        double banda5=calibration5[0]+((calibration5[1]-calibration5[0])/255.0f)*pixel5;
        double banda6=calibration6[0]+((calibration6[1]-calibration6[0])/255.0f)*pixel6;
        double banda7=calibration7[0]+((calibration7[1]-calibration7[0])/255.0f)*pixel7;
        double bandaRefletida1=(pi*banda1)/(calibration1[2]*cosZ*dr);
        sumBandas += parameterAlbedo[0]*bandaRefletida1;
        double bandaRefletida2=(pi*banda2)/(calibration2[2]*cosZ*dr);
        sumBandas += parameterAlbedo[1]*bandaRefletida2;
        double bandaRefletida3=(pi*banda3)/(calibration3[2]*cosZ*dr);
        sumBandas += parameterAlbedo[2]*bandaRefletida3;
        double bandaRefletida4=(pi*banda4)/(calibration4[2]*cosZ*dr);
        sumBandas += parameterAlbedo[3]*bandaRefletida4;
        double bandaRefletida5=(pi*banda5)/(calibration5[2]*cosZ*dr);
        sumBandas += parameterAlbedo[4]*bandaRefletida5;
        double bandaRefletida6=(pi*banda6)/(calibration6[2]*cosZ*dr);
        sumBandas += parameterAlbedo[5]*bandaRefletida6;
        double bandaRefletida7=(pi*banda7)/(calibration7[2]*cosZ*dr);
        sumBandas += parameterAlbedo[6]*bandaRefletida7;
        *albedo = (sumBandas-reflectanciaAtmosfera)/(transmissividade*transmissividade);
        *NDVI = (bandaRefletida4-bandaRefletida3)/(bandaRefletida4+bandaRefletida3);
        *SAVI = ((1.0f+L)*(bandaRefletida4-bandaRefletida3))/(L+bandaRefletida4+bandaRefletida3);
        *IAF = (-logf((0.69f- *SAVI)/0.59f)/0.91f);
        *emissividadeNB = 0.97f+0.0033f* *IAF;
        *emissivity = 0.95f+0.01f* *IAF;
        *Ts = K2/logf((( *emissividadeNB*K1)/banda6)+1.0f);
        *LWd =  *emissivity*StefanBoltzman*(powf( *Ts,4.0f));
        *Rn = ((1.0f- *albedo)*SWd)+( *emissivity*(LWdAtm)- *LWd);
    }

    __global__ void execute(
        double * pixel2,
        double * pixel3,
        double * pixel4,
        double * pixel5,
        double * pixel1,
        double * pixel7,
        double * pixel6,

        double * albedo,
        double * NDVI,
        double * SAVI,
        double * IAF,
        double * emissividadeNB,
        double * emissivity,
        double * Ts,
        double * LWd,
        double * Rn,
        int size){
        int idx = blockIdx.x*blockDim.x + threadIdx.x;
        if(idx < size){
            execute_sub(
                pixel2[idx],
                pixel3[idx],
                pixel4[idx],
                pixel5[idx],
                pixel1[idx],
                pixel7[idx],
                pixel6[idx],
                (albedo+idx),
                (NDVI+idx),
                (SAVI+idx),
                (IAF+idx),
                (emissividadeNB+idx),
                (emissivity+idx),
                (Ts+idx),
                (LWd+idx),
                (Rn+idx)
            );
        }
    }
}

